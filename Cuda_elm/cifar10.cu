//Libraries for cuda runtime
#include <hipblas.h>
#include <hiprand.h>

//Standard C libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "Display_Matrix.h"
#include "Random_Matrix.h"
#include "Inverse_Matrix.h"
#include "PInverse_Matrix.h"
#include "Load_Matrix.h"
#include "train.h"
#include "test.h"

//Main function
int main()
{	
	
	float *Mat_f;
	float *beta;
	float *op_matrix;
	unsigned long training_samples = 50000;
	unsigned long testing_samples = 10000;
	unsigned long ip_num  = 64;
	unsigned long op_num = 10;
	unsigned long hid_num = 20;
	Mat_f= (float *)malloc(ip_num* hid_num* sizeof(float));
	beta= (float *)malloc(hid_num* op_num* sizeof(float));
	op_matrix= (float *)malloc(testing_samples* op_num* sizeof(float));
	 
	
	float *X_Train = (float *)malloc(training_samples	* ip_num	* sizeof(float));
	float *Y_Train = (float *)malloc(training_samples	* op_num	* sizeof(float));
	float *X_Test = (float *)malloc(testing_samples 	* ip_num	* sizeof(float));
	float *Y_Test = (float *)malloc(testing_samples	* op_num	* sizeof(float));
	
	Import_Fromfile(X_Train,"features_cifar10/train_features.csv");
	Import_Fromfile(Y_Train,"features_cifar10/train_labels.csv");
	Import_Fromfile(X_Test,"features_cifar10/test_features.csv");
	Import_Fromfile(Y_Test,"features_cifar10/test_labels.csv");
	

	//// Calling a training function of ELM
	Train_elm(X_Train,Y_Train,Mat_f,beta,ip_num,hid_num,op_num,training_samples);
	
	//// Calling a testing function of ELM
	Test_elm(X_Test,Y_Test,Mat_f,beta,op_matrix,ip_num,hid_num,op_num,testing_samples);
	
	/// Output Matrix and Accuracy
	Display_Matrix(op_matrix,Y_Test,testing_samples,op_num);
	printf("\n");
	
	return 0;
}